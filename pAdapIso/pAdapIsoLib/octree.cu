#ifndef _OCTREE_KERNEL_H_
#define _OCTREE_KERNEL_H_

#include <stdio.h>

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
testKernel( float* g_idata, float* g_odata) 
{
  // shared memory
  // the size is determined by the host application
  extern  __shared__  float sdata[];

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;

  // read in input data from global memory
  sdata[tid] = g_idata[tid];
  __syncthreads();

  // perform some computations
  sdata[tid] = (float) num_threads * sdata[tid];
  __syncthreads();

  // write data to global memory
  g_odata[tid] = sdata[tid];
}

#endif // #ifndef _TEMPLATE_KERNEL_H_
